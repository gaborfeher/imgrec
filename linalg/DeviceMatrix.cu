#include "DeviceMatrix.h"

#include "HostMatrix.h"

#include <hip/hip_runtime.h>

#include <iostream>

DeviceMatrix::DeviceMatrix(const HostMatrix& src) :
    rows_(src.rows_),
    cols_(src.cols_),
    size_(src.size_) {
  hipMalloc(&data_, size_ * sizeof(float)); 
  hipMemcpy(data_, src.data_.get(), size_ * sizeof(float), hipMemcpyHostToDevice);
}

DeviceMatrix::~DeviceMatrix() {
  hipFree(data_);
}
