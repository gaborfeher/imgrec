#include "DeviceMatrix.h"

#include <iostream>

#include <hip/hip_runtime.h>

#include "HostMatrix.h"

DeviceMatrix::DeviceMatrix(const HostMatrix& src) :
    BaseMatrix(src.rows_, src.cols_) {
  float* data;
  hipMalloc(&data, size_ * sizeof(float));
  data_.reset(data, hipFree);
  hipMemcpy(
      data_.get(),
      src.data_.get(),
      size_ * sizeof(float),
      hipMemcpyHostToDevice);
}

