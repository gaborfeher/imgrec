#include "DeviceMatrix.h"

#include "HostMatrix.h"

#include <hip/hip_runtime.h>

#include <iostream>

DeviceMatrix::DeviceMatrix(const HostMatrix& src) :
    rows_(src.rows_),
    cols_(src.cols_),
    size_(src.size_) {
  float* data;
  hipMalloc(&data, size_ * sizeof(float));
  data_.reset(data, hipFree);
  hipMemcpy(
      data_.get(),
      src.data_.get(),
      size_ * sizeof(float),
      hipMemcpyHostToDevice);
}

